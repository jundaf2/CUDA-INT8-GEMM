// System includes
#include <assert.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <vector>
#include <random>
#include <algorithm>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cpu_gemm.h"
#include "util.h"
#include "gemm_i8.cuh"

using namespace std;


template <bool use_tcu, typename T>
void GEMMI8(hipStream_t stream, 
            const int8_t *A, const int8_t *B, T *C,
            int M, int N, int K,
            bool transA, bool transB, bool transC) 
{
  constexpr int BLOCK_M = 128;
  constexpr int BLOCK_N = 128;
  constexpr int BLOCK_K = 32;
  constexpr int WARP_M = 32;
  constexpr int WARP_N = 32;
  constexpr int WARP_SIZE = 32;

  dim3 block((BLOCK_M / WARP_M) * (BLOCK_N / WARP_N) * WARP_SIZE, 1, 1);   
  dim3 grid((N + BLOCK_N - 1) / BLOCK_N, (M + BLOCK_M - 1) / BLOCK_M);  
  
  if(transA==GEMM_OP_T && transB==GEMM_OP_N && transC==GEMM_OP_T)
    wmma_kernel::GEMMI8TCU<BLOCK_M, BLOCK_N, BLOCK_K, WARP_M, WARP_N, 2, GEMM_OP_T, GEMM_OP_N, GEMM_OP_T><<<grid, block, 0, stream>>>(A, B, C, M, N, K);
  
}


template <typename T>
class GEMM {
public:
  GEMM(bool use_tcu, int m, int n, int k, bool transa, bool transb, bool transc) {
    this->use_tcu = use_tcu;

    this->M = m;
    this->N = n;
    this->K = k;

    this->len_a = M*K;
    this->len_b = N*K;
    this->len_c = M*N;

    this->trans_a = transa;
    this->trans_b = transb;
    this->trans_c = transc;

    cout << "compute type=int32" << ", "
          << "input data type=int8" << ", "
          << "output data type=" << (std::is_same<T,int8_t>::value ? "int8" : "int32") << ", "
          << "use_tcu=" << use_tcu << ", "
          << "M=" << m << ", "
          << "N=" << n << ", "
          << "K=" << k
          << endl;

    generateTestData();
  }

  ~GEMM()  = default;

  void generateTestData() {
    
    const auto random_seed = 2023;
    std::mt19937 generator(static_cast<unsigned int>(random_seed));

    h_mat_A = vector<int8_t>(len_a, 0);
    h_mat_B = vector<int8_t>(len_b, 0);
    h_mat_C = vector<T>(len_c, 0);
    h_mat_C_ref = vector<T>(len_c, 0);

    std::uniform_int_distribution<> uniform_char_distribution(CHAR_MIN, CHAR_MAX);

    auto rand_gen = std::bind(uniform_char_distribution, generator);
    auto const_gen = []() { return 1; };
    auto pattern_gen = []() { static int i = 0; return (i++)/32%64; };

    generate_n(h_mat_A.begin(), len_a, rand_gen);
    generate_n(h_mat_B.begin(), len_b, rand_gen);

  }

public:
  void testGEMM() {
    hipStream_t stream;
    ASSERT_CUDA(hipStreamCreate(&stream));

    // CPU reference
    {
      cpuGEMM<float, float, int8_t, T>(
          h_mat_A.data(), h_mat_B.data(), h_mat_C_ref.data(), M, N, K,
          len_a, len_b, len_c, 1, static_cast<float>(1), static_cast<float>(0), 
          GEMM_OP_T, GEMM_OP_N, GEMM_OP_T);
    }

    ASSERT_CUDA(hipMalloc(&d_mat_A, len_a * sizeof(int8_t))); 
    ASSERT_CUDA(hipMalloc(&d_mat_B, len_b * sizeof(int8_t)));
    ASSERT_CUDA(hipMalloc(&d_mat_C, len_c * sizeof(T)));

    ASSERT_CUDA(hipMemcpy(d_mat_A, h_mat_A.data(), len_a * sizeof(int8_t), hipMemcpyHostToDevice)); 
    ASSERT_CUDA(hipMemcpy(d_mat_B, h_mat_B.data(), len_b * sizeof(int8_t), hipMemcpyHostToDevice));
    ASSERT_CUDA(hipMemset(d_mat_C, 0, len_c * sizeof(T)));

    // warp up the device
    {  
      if(use_tcu) GEMMI8<true, T>(stream, d_mat_A, d_mat_B, d_mat_C, M, N, K, trans_a, trans_b, trans_c);
    }

    // time it
    float milliseconds = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, stream);

    {  
      if(use_tcu) GEMMI8<true, T>(stream, d_mat_A, d_mat_B, d_mat_C, M, N, K, trans_a, trans_b, trans_c);
    }

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds , start, stop);
    
    double   flops = static_cast<double>(M)*static_cast<double>(N)*static_cast<double>(K)*2*1.0;
    double   tetraFlops = (flops * 1.0e-12f) / (milliseconds  / 1000.0f);
    double   bandWidth = (static_cast<double>(len_a+len_b)*sizeof(int8_t)+static_cast<double>(len_c)*sizeof(T)) / (milliseconds  * 1000 * 1000);
    printf("\033[31;47m INT8 GEMM took %.6f ms, %.2f T OP/s, %.2f GB/s \033[0m\n", milliseconds , tetraFlops, bandWidth);
    ASSERT_CUDA(hipDeviceSynchronize());
    ASSERT_CUDA(hipEventDestroy(start));
    ASSERT_CUDA(hipEventDestroy(stop));
    
    ASSERT_CUDA(hipMemcpy(h_mat_C.data(), d_mat_C, len_c * sizeof(T), hipMemcpyDeviceToHost));
    
    ASSERT_CUDA(hipFree(d_mat_A));
    ASSERT_CUDA(hipFree(d_mat_B));
    ASSERT_CUDA(hipFree(d_mat_C));
    ASSERT_CUDA(hipStreamDestroy(stream));

    print_vec(h_mat_C.data(), "h_mat_C: ", 0, 32, N);
    print_vec(h_mat_C_ref.data(), "h_mat_C_ref: ", 0, 32, N);

    if(h_mat_C == h_mat_C_ref) {
      cout << "test passed !" << endl;
    } else {
      cout << "test failed !" << endl;
    }
  }

protected:

  bool use_tcu;
  int M, N, K;
  long long int len_a, len_b, len_c;
  bool trans_a, trans_b, trans_c;

  vector<int8_t> h_mat_A;
  vector<int8_t> h_mat_B;
  vector<T> h_mat_C;
  vector<T> h_mat_C_ref;

  int8_t *d_mat_A;
  int8_t *d_mat_B;
  T *d_mat_C;
};


int main(int argc, char **argv) {
  // minimum setting
  int M = 256;
  int N = 256;
  int K = 32;

  bool trans_a = GEMM_OP_T;
  bool trans_b = GEMM_OP_N;
  bool trans_c = GEMM_OP_T;

  bool use_tcu = true;

  if(argc > 1) {
    M = atoi(argv[1]);
  }
  if(argc > 2) {
    N = atoi(argv[2]);
  }
  if(argc > 3) {
    K = atoi(argv[3]);
  }
  if(argc > 4) {
    trans_a = atoi(argv[4]);
  }
  if(argc > 5) {
    trans_b = atoi(argv[5]);
  }
  if(argc > 6) {
    trans_c = atoi(argv[6]);
  }
  if(argc > 7) {
    use_tcu = atoi(argv[7]);
  }


  GEMM<int8_t> gemm(use_tcu, M, N, K, trans_a, trans_b, trans_c);
  gemm.testGEMM();


  return 0;
}