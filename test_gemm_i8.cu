// System includes
#include <assert.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <vector>
#include <random>
#include <algorithm>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cpu_gemm.h"
#include "util.h"
#include "gemm_i8.cuh"

using namespace std;


template <bool use_tcu>
void GEMMI8(hipStream_t stream, 
            const int8_t *A, const int8_t *B, int8_t *C,
            int M, int N, int K,
            bool transA, bool transB, bool transC) 
{
  constexpr int BLOCK_M = 256;
  constexpr int BLOCK_N = 256;
  constexpr int BLOCK_K = 32;
  constexpr int WARP_M = 64;
  constexpr int WARP_N = 64;
  constexpr int WARP_SIZE = 32;

  dim3 block((BLOCK_M / WARP_M) * (BLOCK_N / WARP_N) * WARP_SIZE, 1, 1);   
  dim3 grid((N + BLOCK_N - 1) / BLOCK_N / 1, (M + BLOCK_M - 1) / BLOCK_M);  
  
  if(transA==GEMM_OP_T && transB==GEMM_OP_N && transC==GEMM_OP_T)
    kernel::GEMMI8TCU<BLOCK_M, BLOCK_N, BLOCK_K, WARP_M, WARP_N, 2, GEMM_OP_T, GEMM_OP_N, GEMM_OP_T><<<grid, block, 0, stream>>>(A, B, C, M, N, K);
  
}


class GEMM {
public:
  GEMM(bool use_tcu, int m, int n, int k, bool transa, bool transb, bool transc) {
    this->use_tcu = use_tcu;
    this->M = m;
    this->N = n;
    this->K = k;

    this->len_a = M*K;
    this->len_b = N*K;
    this->len_c = M*N;

    this->trans_a = transa;
    this->trans_b = transb;
    this->trans_c = transc;

    cout << "compute type=int32" << ", "
          << "data type=int8" << ", "
          << "use_tcu=" << use_tcu << ", "
          << "M=" << m << ", "
          << "N=" << n << ", "
          << "K=" << k
          << endl;

    generateTestData();
  }

  ~GEMM()  = default;

  void generateTestData() {
    
    const auto random_seed = 2023;
    std::mt19937 generator(static_cast<unsigned int>(random_seed));

    h_mat_A = vector<int8_t>(len_a, 0);
    h_mat_B = vector<int8_t>(len_b, 0);
    h_mat_C = vector<int8_t>(len_c, 0);
    h_mat_C_ref = vector<int8_t>(len_c, 0);

    std::uniform_int_distribution<> uniform_char_distribution(CHAR_MIN, CHAR_MAX);

    auto rand_gen = std::bind(uniform_char_distribution, generator);
    auto const_gen = []() { return 1; };

    generate_n(h_mat_A.begin(), len_a, const_gen);
    generate_n(h_mat_B.begin(), len_b, const_gen);

  }

public:
  void testGEMM() {
    hipStream_t stream;
    ASSERT_CUDA(hipStreamCreate(&stream));

    // CPU reference
    {
      cpuGEMM<float, float, int8_t, int8_t>(
          h_mat_A.data(), h_mat_B.data(), h_mat_C_ref.data(), M, N, K,
          len_a, len_b, len_c, 1, static_cast<float>(1), static_cast<float>(0), 
          GEMM_OP_T, GEMM_OP_N, GEMM_OP_T);
    }

    ASSERT_CUDA(hipMalloc(&d_mat_A, len_a * sizeof(int8_t))); 
    ASSERT_CUDA(hipMalloc(&d_mat_B, len_b * sizeof(int8_t)));
    ASSERT_CUDA(hipMalloc(&d_mat_C, len_c * sizeof(int8_t)));

    ASSERT_CUDA(hipMemcpy(d_mat_A, h_mat_A.data(), len_a * sizeof(int8_t), hipMemcpyHostToDevice)); 
    ASSERT_CUDA(hipMemcpy(d_mat_B, h_mat_B.data(), len_b * sizeof(int8_t), hipMemcpyHostToDevice));
    ASSERT_CUDA(hipMemset(d_mat_C, 0, len_c * sizeof(int8_t)));

    // warp up the device

    // time it
    float milliseconds = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, stream);

    {  
      if(use_tcu) GEMMI8<true>(stream, d_mat_A, d_mat_B, d_mat_C, M, N, K, trans_a, trans_b, trans_c);
    }

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds , start, stop);
    
    double   flops = (double)(M*N*K*2)*1.0;
    double   gigaFlops = (flops * 1.0e-9f) / (milliseconds  / 1000.0f);
    double   bandWidth = (double)(len_a+len_b+len_c)*sizeof(int8_t) / (milliseconds  * 1000 * 1000);
    printf("\033[31;47m INT8 GEMM took %.3f ms, %.2f GFlop/s, %.2f GB/s \033[0m\n", milliseconds , gigaFlops, bandWidth);
    ASSERT_CUDA(hipDeviceSynchronize());
    ASSERT_CUDA(hipEventDestroy(start));
    ASSERT_CUDA(hipEventDestroy(stop));
    ASSERT_CUDA(hipMemcpy(h_mat_C.data(), d_mat_C, len_c * sizeof(int8_t), hipMemcpyDeviceToHost));
    
    ASSERT_CUDA(hipFree(d_mat_A));
    ASSERT_CUDA(hipFree(d_mat_B));
    ASSERT_CUDA(hipFree(d_mat_C));
    ASSERT_CUDA(hipStreamDestroy(stream));

    print_vec(h_mat_C.data(), "h_mat_C: ", 0, N, N);
    print_vec(h_mat_C_ref.data(), "h_mat_C_ref: ", 0, N, N);

    if(h_mat_C == h_mat_C_ref) {
      cout << "test passed !" << endl;
    } else {
      cout << "test failed !" << endl;
    }
  }

protected:

  bool use_tcu;
  int M, N, K;
  int len_a, len_b, len_c;
  bool trans_a, trans_b, trans_c;

  vector<int8_t> h_mat_A;
  vector<int8_t> h_mat_B;
  vector<int8_t> h_mat_C;
  vector<int8_t> h_mat_C_ref;

  int8_t *d_mat_A;
  int8_t *d_mat_B;
  int8_t *d_mat_C;
};


int main(int argc, char **argv) {
  int M = 256;
  int N = 256;
  int K = 32;

  bool trans_a = GEMM_OP_T;
  bool trans_b = GEMM_OP_N;
  bool trans_c = GEMM_OP_T;

  bool use_tcu = true;

  if(argc > 1) {
    M = atoi(argv[1]);
  }
  if(argc > 2) {
    N = atoi(argv[2]);
  }
  if(argc > 3) {
    K = atoi(argv[3]);
  }
  if(argc > 4) {
    trans_a = atoi(argv[4]);
  }
  if(argc > 5) {
    trans_b = atoi(argv[5]);
  }
  if(argc > 6) {
    trans_c = atoi(argv[6]);
  }



  GEMM gemm(use_tcu, M, N, K, trans_a, trans_b, trans_c);
  gemm.testGEMM();

  return 0;
}